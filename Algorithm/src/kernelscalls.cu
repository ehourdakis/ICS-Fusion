#include "hip/hip_runtime.h"
#include"kernelscalls.h"
#include"kernels.h"
#include <thrust/device_vector.h>
#include "constant_parameters.h"
#include<iostream>

sMatrix6 calculatePoint2PointCov(const float3 *vert,
                                 int vertSize,
                                 const float3 *prevVert,
                                 int prevVertSize,
                                 const int2 *corresp,
                                 size_t correspSize,
                                 const sMatrix4 &tf,
                                 const kparams_t &params)
{
    float3 *vertGpu;
    hipMalloc(&vertGpu,vertSize*sizeof(float3));
    hipMemcpy(vertGpu,vert,vertSize*sizeof(float3),hipMemcpyHostToDevice);

    float3 *prevVertGpu;
    hipMalloc(&prevVertGpu,prevVertSize*sizeof(float3));
    hipMemcpy(prevVertGpu,prevVert,prevVertSize*sizeof(float3),hipMemcpyHostToDevice);

    int2 *correspGpu;
    int err=hipMalloc(&correspGpu,correspSize*sizeof(int2));
    hipMemcpy(correspGpu,corresp,correspSize*sizeof(int2),hipMemcpyHostToDevice);

    sMatrix6 *covData;
    hipMalloc(&covData,correspSize*sizeof(sMatrix6));
    
    point2PointCovFirstTerm<<<(correspSize+256)/256, 256>>>(vertGpu,
                                                            vertSize,
                                                            prevVertGpu,
                                                            prevVertSize,
                                                            correspGpu,
                                                            correspSize,
                                                            tf,
                                                            covData,
                                                            params.cov_big);
    sMatrix6 initMat;
    for(int i=0;i<36;i++)
        initMat.data[i]=0.0;
    
    hipDeviceSynchronize();
    
    
    
    thrust::device_ptr<sMatrix6> cov_ptr(covData);
    sMatrix6 d2J_dX2 = thrust::reduce(cov_ptr, cov_ptr+correspSize, initMat, thrust::plus<sMatrix6>());

    float cov_z=1;
    point2PointCovSecondTerm<<<(correspSize+256)/256, 256>>>(vertGpu,
                                                            vertSize,
                                                            prevVertGpu,
                                                            prevVertSize,
                                                            correspGpu,
                                                            correspSize,
                                                            tf,
                                                            cov_z,
                                                            covData,
                                                            params.cov_big);
    hipDeviceSynchronize();    
    sMatrix6 covSecondTerm = thrust::reduce(cov_ptr, cov_ptr+correspSize, initMat, thrust::plus<sMatrix6>());


    sMatrix6 d2J_dX2inv=inverse(d2J_dX2);
    sMatrix6 tmp=d2J_dX2inv * covSecondTerm;
    sMatrix6 icpCov= tmp * d2J_dX2inv;
    
    //make sure that covariance matrix is symetric.
    //small asymetries may occur due to numerical stability
    sMatrix6 ret;
    for(int i=0;i<6;i++)
    {
        for(int j=0;j<6;j++)
        {
            //eliminate NaN values
            if(icpCov(i,j)!=icpCov(i,j))
            {
                icpCov(i,j)=params.cov_big;
            }
            if(icpCov(j,i)!=icpCov(j,i))
            {
                icpCov(j,i)=params.cov_big;
            }
            float val=( icpCov(i,j) + icpCov(j,i))/2;
            ret(i,j)=val;
            ret(j,i)=val;

        }
    }
    
    hipFree(vertGpu);
    hipFree(prevVertGpu);
    hipFree(correspGpu);
    hipFree(covData);
    
    return ret;
}
