#include "hip/hip_runtime.h"
#include"kernelscalls.h"
#include"kernels.h"

sMatrix6 calculatePoint2PointCov(float3 *vert,
                                 int vertSize,
                                 float3 *prevVert,
                                 int prevVertSize,
                                 int2 *corresp,
                                 int correspSize,
                                 sMatrix4 tf)
{
    sMatrix6 ret;

    float3 *vertGpu;
    hipMalloc(&vertGpu,vertSize*sizeof(float3));
    hipMemcpy(vertGpu,vert,vertSize*sizeof(float3),hipMemcpyHostToDevice);

    float3 *prevVertGpu;
    hipMalloc(&prevVertGpu,prevVertSize*sizeof(float3));
    hipMemcpy(prevVertGpu,prevVert,prevVertSize*sizeof(float3),hipMemcpyHostToDevice);

    int2 *correspGpu;
    hipMalloc(&correspGpu,correspSize*sizeof(int2));
    hipMemcpy(correspGpu,corresp,correspSize*sizeof(int2),hipMemcpyHostToDevice);

//    sMatrix6 covData;
//    point2PointCovFirstTerm<<<grid, imageBlock>>>(vertGpu,
//                                                  vertSize,
//                                                  prevVertGpu,
//                                                  prevVertSize,
//                                                  corresp,
//                                                  correspSize,
//                                                  tf,
//                                                  covData);


    hipFree(vertGpu);
    hipFree(prevVertGpu);
    hipFree(correspGpu);

    return ret;


    /*
    sMatrix4 currPose=pose;
    sMatrix4 invPrevPose=inverse(oldPose);
    sMatrix4 delta=invPrevPose*currPose;

    Matrix4 projectedReference = camMatrix*inverse(Matrix4(&raycastPose));
    dim3 grid=divup(make_uint2(params.inputSize.x,params.inputSize.y),imageBlock );

    sMatrix6 initMat;
    for(int i=0;i<36;i++)
        initMat.data[i]=0.0;


    icpCovarianceFirstTerm<<<grid, imageBlock>>>(inputVertex[0],
                                                vertex,
                                                normal,
                                                reduction,
                                                covData,
                                                trackPose,
                                                projectedReference,
                                                delta);

    hipDeviceSynchronize();
    size_t size=covData.size.x*covData.size.y;
    thrust::device_ptr<sMatrix6> cov_ptr(covData.data());
    sMatrix6 d2J_dX2 = thrust::reduce(cov_ptr, cov_ptr+size, initMat, thrust::plus<sMatrix6>());

    icpCovarianceSecondTerm<<<grid, imageBlock>>>(inputVertex[0],
                                                  vertex,
                                                  normal,
                                                  reduction,
                                                  covData,
                                                  trackPose,
                                                  projectedReference,
                                                  delta,
                                                  1.0);
    hipDeviceSynchronize();
    sMatrix6 covSecondTerm = thrust::reduce(cov_ptr, cov_ptr+size, initMat, thrust::plus<sMatrix6>());


    sMatrix6 d2J_dX2inv=inverse(d2J_dX2);
    sMatrix6 tmp=d2J_dX2inv * covSecondTerm;
    sMatrix6 icpCov= tmp * d2J_dX2inv;

    //make sure that covariance matrix is symetric.
    //small asymetries may occur due to numerical stability
    sMatrix6 ret;
    for(int i=0;i<6;i++)
    {
        for(int j=0;j<6;j++)
        {
            //eliminate NaN values
            if(icpCov(i,j)!=icpCov(i,j))
            {
                icpCov(i,j)=cov_big;
            }
            if(icpCov(j,i)!=icpCov(j,i))
            {
                icpCov(j,i)=cov_big;
            }
            float val=( icpCov(i,j) + icpCov(j,i))/2;
            ret(i,j)=val;
            ret(j,i)=val;

        }
    }
    */

}
